#include "hip/hip_runtime.h"
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/opencv.hpp>
//#include "utils.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <string>

cv::Mat imageRGBA;
cv::Mat imageOut;

uchar4 *d_rgbaImage__;
uchar4 *d_blurImage__;

size_t numRows() { return imageRGBA.rows; }
size_t numCols() { return imageRGBA.cols; }

//return types are void since any internal error will be handled by quitting
//no point in returning error codes...
//returns a pointer to an RGBA version of the input image
//and a pointer to the single channel grey-scale output
//on both the host and device
void preProcess(uchar4 **inputImage, uchar4 **blurImage,
                uchar4 **d_rgbaImage, uchar4 **d_blurImage,
                const std::string &filename) {
  //make sure the context initializes ok
  hipFree(0);//checkCudaErrors(hipFree(0));

  cv::Mat image;
  image = cv::imread(filename.c_str(), CV_LOAD_IMAGE_COLOR);
  if (image.empty()) {
    std::cerr << "Couldn't open file: " << filename << std::endl;
    exit(1);
  }

  cv::cvtColor(image, imageRGBA, CV_BGR2RGBA);

  //allocate memory for the output
  imageOut.create(image.rows, image.cols, CV_32FC4); //CV_8UC1

  //This shouldn't ever happen given the way the images are created
  //at least based upon my limited understanding of OpenCV, but better to check
  if (!imageRGBA.isContinuous() || !imageOut.isContinuous()) {
    std::cerr << "Images aren't continuous!! Exiting." << std::endl;
    exit(1);
  }

  *inputImage = (uchar4 *)imageRGBA.ptr<unsigned char>(0);
  *blurImage  = (uchar4 *)imageOut.ptr<unsigned char>(0);

  const size_t numPixels = numRows() * numCols();
  //allocate memory on the device for both input and output
  hipMalloc(d_rgbaImage, sizeof(uchar4) * numPixels); // checkCudaErrors
  hipMalloc(d_blurImage, sizeof(uchar4) * numPixels); // checkCudaErrors
  hipMemset(*d_blurImage, 0, numPixels * sizeof(uchar4)); //make sure no memory is left laying around; checkCudaErrors

  //copy input array to the GPU
  hipMemcpy(*d_rgbaImage, *inputImage, sizeof(uchar4) * numPixels, hipMemcpyHostToDevice); // checkCudaErrors

  d_rgbaImage__ = *d_rgbaImage;
  d_blurImage__ = *d_blurImage;
}

void postProcess(const std::string& output_file) {
  const int numPixels = numRows() * numCols();
  //copy the output back to the host
  hipMemcpy(imageOut.ptr<uchar4>(0), d_blurImage__, sizeof(uchar4) * numPixels, hipMemcpyDeviceToHost); // checkCudaErrors

  //output the image
  cv::Mat imageOut2;
  imageOut2.create(imageOut.rows, imageOut.cols, CV_8UC4);
  cv::cvtColor(imageOut, imageOut2, CV_RGBA2BGR);
  cv::imwrite(output_file.c_str(), imageOut2);

  //cleanup
  hipFree(d_rgbaImage__);
  hipFree(d_blurImage__);
}
