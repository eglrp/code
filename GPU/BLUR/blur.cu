// Homework 1
// Color to Greyscale Conversion

//A common way to represent color images is known as RGBA - the color
//is specified by how much Red, Grean and Blue is in it.
//The 'A' stands for Alpha and is used for transparency, it will be
//ignored in this homework.

//Each channel Red, Blue, Green and Alpha is represented by one byte.
//Since we are using one byte for each color there are 256 different
//possible values for each color.  This means we use 4 bytes per pixel.

//Greyscale images are represented by a single intensity value per pixel
//which is one byte in size.

//To convert an image from color to grayscale one simple method is to
//set the intensity to the average of the RGB channels.  But we will
//use a more sophisticated method that takes into account how the eye 
//perceives color and weights the channels unequally.

//The eye responds most strongly to green followed by red and then blue.
//The NTSC (National Television System Committee) recommends the following
//formula for color to greyscale conversion:

//I = .299f * R + .587f * G + .114f * B

//Notice the trailing f's on the numbers which indicate that they are 
//single precision floating point constants and not double precision
//constants.

//You should fill in the kernel as well as set the block and grid sizes
//so that the entire image is processed.

//#include "image_IO.cu"
//#include "utils.h"

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#define blks 32;

__global__
void blur_image(const uchar4* const rgbaImage,
                       uchar4* greyImage,
                       int numRows, int numCols)
{
  //TODO
  //Fill in the kernel to convert from color to greyscale
  //the mapping from components of a uchar4 to RGBA is:
  // .x -> R ; .y -> G ; .z -> B ; .w -> A
  //
  //The output (greyImage) at each pixel should be the result of
  //applying the formula: output = .299f * R + .587f * G + .114f * B;
  //Note: We will be ignoring the alpha channel for this conversion

  //First create a mapping from the 2D block and grid locations
  //to an absolute 2D location in the image, then use that to
  //calculate a 1D offset
    int idx, idxc, npt=0;
    int ix, x = blockIdx.x * 32 + threadIdx.x;
    if( x >= numCols ) return;
    int iy, y = blockIdx.y * 16 + threadIdx.y;
    if( y >= numRows ) return;
    uchar4 rgbain;
    idxc = x + y * numCols;
    for(ix=x-0; ix<x+1; ix++) {
       if( ix<0 || ix>numCols ) continue;
       for(iy=y-0; iy<y+1; iy++) {
          if( iy<0 || iy>=numRows ) continue;
          idx = ix + iy * numCols;
          rgbain = rgbaImage[idx];
          greyImage[idxc].x += rgbain.x;
          greyImage[idxc].y += rgbain.y; 
          greyImage[idxc].z += rgbain.z;
	  npt++;
       }
    }
    //greyImage[idxc].x /= npt;
    //greyImage[idxc].y /= npt; 
    //greyImage[idxc].z /= npt; 
    //greyImage[idxc].w = rgbaImage[idxc].w;
    greyImage[idxc] = rgbaImage[idxc];
}

void your_rgba_to_greyscale(const uchar4 * const h_rgbaImage, uchar4 * const d_rgbaImage,
                            uchar4* d_greyImage, size_t numRows, size_t numCols)
{
  //You must fill in the correct sizes for the blockSize and gridSize
  //currently only one block with one thread is being launched
  const dim3 blockSize(32, 16, 1);  //TODO
  const dim3 gridSize( numCols/32+1, numRows/16+1, 1);  //TODO
  blur_image<<<gridSize, blockSize>>>(d_rgbaImage, d_greyImage, numRows, numCols);
 
  hipDeviceSynchronize(); //checkCudaErrors(cudaGetLastError());
}
